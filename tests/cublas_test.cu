#include <iostream>
#include <hipblas.h>
#include <hipblas.h>

template <class T>
hipblasStatus_t gemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
template <>
hipblasStatus_t gemm<float >(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {
	return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
template <>
hipblasStatus_t gemm<double>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const double *alpha, const double *A, int lda,
                           const double *B, int ldb, const double *beta, double *C,
                           int ldc) {
	return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
template <>
hipblasStatus_t gemm<half  >(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const half *alpha, const half *A, int lda,
                           const half *B, int ldb, const half *beta, half *C,
                           int ldc) {
	return hipblasHgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

template <class T>
void gemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = static_cast<T>(1);
	const auto beta  = static_cast<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm<T>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

int main(){
	gemm_test<double>();
	gemm_test<float >();
	gemm_test<half  >();
}
