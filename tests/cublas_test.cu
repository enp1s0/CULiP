#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hipblas.h>
#include <CULiP/cublas.hpp>

struct op_gemm;
struct op_gemmEx;

template <class T, class Op>
hipblasStatus_t gemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##gemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_GEMM(H, half);
GEMM_OP_GEMM(S, float);
GEMM_OP_GEMM(D, double);
GEMM_OP_GEMM(C, hipComplex);
GEMM_OP_GEMM(Z, hipDoubleComplex);

#define GEMM_OP_GEMM_EX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return hipblasGemmEx(handle, transa, transb, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_OP_GEMM_EX(HIP_R_16F, half);
GEMM_OP_GEMM_EX(HIP_R_32F, float);
GEMM_OP_GEMM_EX(HIP_R_64F, double);
GEMM_OP_GEMM_EX(HIP_C_32F, hipComplex);
GEMM_OP_GEMM_EX(HIP_C_64F, hipDoubleComplex);

// -------------
// GemmBatched
// -------------
template <class T, class Op>
hipblasStatus_t gemm_batched(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A[], int lda,
                           const T *B[], int ldb, const T *beta, T *C[],
                           int ldc, int batchCount);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_BATCHED_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm_batched<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A[], int lda,\
                           const type *B[], int ldb, const type *beta, type *C[],\
                           int ldc, int batchCount) {\
	return cublas##short_type##gemmBatched(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, batchCount);\
}
GEMM_BATCHED_OP_GEMM(H, half);
GEMM_BATCHED_OP_GEMM(S, float);
GEMM_BATCHED_OP_GEMM(D, double);
GEMM_BATCHED_OP_GEMM(C, hipComplex);
GEMM_BATCHED_OP_GEMM(Z, hipDoubleComplex);
// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
#define GEMM_BATCHED_OP_GEMMEX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm_batched<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A[], int lda,\
                           const type *B[], int ldb, const type *beta, type *C[],\
                           int ldc, int batchCount) {\
	return hipblasGemmBatchedEx(handle, transa, transb, m, n, k, alpha, reinterpret_cast<const void**>(A), cuda_data_type, lda, reinterpret_cast<const void**>(B), cuda_data_type, ldb, beta, reinterpret_cast<void**>(C), cuda_data_type, ldc, batchCount, cuda_data_type, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_BATCHED_OP_GEMMEX(HIP_R_16F, half);
GEMM_BATCHED_OP_GEMMEX(HIP_R_32F, float);
GEMM_BATCHED_OP_GEMMEX(HIP_R_64F, double);
GEMM_BATCHED_OP_GEMMEX(HIP_C_32F, hipComplex);
GEMM_BATCHED_OP_GEMMEX(HIP_C_64F, hipDoubleComplex);


template <class T>
T convert(const double a) {return static_cast<T>(a);}
template <> hipComplex       convert<hipComplex      >(const double a) {return make_float2(a, 0);}
template <> hipDoubleComplex convert<hipDoubleComplex>(const double a) {return make_double2(a, 0);}

template <class T, class Op>
void gemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

void test_all() {
	gemm_test<double         , op_gemm  >();
	gemm_test<float          , op_gemm  >();
	gemm_test<half           , op_gemm  >();
	gemm_test<hipComplex      , op_gemm  >();
	gemm_test<hipDoubleComplex, op_gemm  >();
	gemm_test<double         , op_gemmEx>();
	gemm_test<float          , op_gemmEx>();
	gemm_test<half           , op_gemmEx>();
	gemm_test<hipComplex      , op_gemmEx>();
	gemm_test<hipDoubleComplex, op_gemmEx>();
}

int main(){
	std::printf("Without profiling\n");
	CULiP_profile_cublas_disable_all();
	test_all();
	std::printf("-------\n");

	std::printf("With profiling\n");
	CULiP_profile_cublas_enable_all();
	test_all();
	std::printf("-------\n");
}
