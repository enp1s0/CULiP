#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hipblas.h>
#include <CULiP/cublas.hpp>

struct op_gemm;
struct op_gemmEx;

template <class T, class Op>
hipblasStatus_t gemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
template <>
hipblasStatus_t gemm<float , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {
	return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
template <>
hipblasStatus_t gemm<double, op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const double *alpha, const double *A, int lda,
                           const double *B, int ldb, const double *beta, double *C,
                           int ldc) {
	return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
template <>
hipblasStatus_t gemm<half  , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const half *alpha, const half *A, int lda,
                           const half *B, int ldb, const half *beta, half *C,
                           int ldc) {
	return hipblasHgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
template <>
hipblasStatus_t gemm<hipComplex, op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const hipComplex *alpha, const hipComplex *A, int lda,
                           const hipComplex *B, int ldb, const hipComplex *beta, hipComplex *C,
                           int ldc) {
	return hipblasCgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
template <>
hipblasStatus_t gemm<float , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {
	return hipblasGemmEx(handle, transa, transb, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
}
template <>
hipblasStatus_t gemm<double, op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const double *alpha, const double *A, int lda,
                           const double *B, int ldb, const double *beta, double *C,
                           int ldc) {
	return hipblasGemmEx(handle, transa, transb, m, n, k, alpha, A, HIP_R_64F, lda, B, HIP_R_64F, ldb, beta, C, HIP_R_64F, ldc, HIP_R_64F, HIPBLAS_GEMM_DEFAULT);
}
template <>
hipblasStatus_t gemm<half  , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const half *alpha, const half *A, int lda,
                           const half *B, int ldb, const half *beta, half *C,
                           int ldc) {
	return hipblasGemmEx(handle, transa, transb, m, n, k, alpha, A, HIP_R_16F, lda, B, HIP_R_16F, ldb, beta, C, HIP_R_16F, ldc, HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
}

template <class T>
T convert(const double a) {return static_cast<T>(a);}
template <> hipComplex       convert<hipComplex      >(const double a) {return make_float2(a, 0);}
template <> hipDoubleComplex convert<hipDoubleComplex>(const double a) {return make_double2(a, 0);}

template <class T, class Op>
void gemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

void test_all() {
	gemm_test<double, op_gemm  >();
	gemm_test<float , op_gemm  >();
	gemm_test<half  , op_gemm  >();
	gemm_test<hipComplex, op_gemm  >();
	gemm_test<double, op_gemmEx>();
	gemm_test<float , op_gemmEx>();
	gemm_test<half  , op_gemmEx>();
}

int main(){
	std::printf("Without profiling\n");
	CULiP_profile_cublas_disable_all();
	test_all();
	std::printf("-------\n");

	std::printf("With profiling\n");
	CULiP_profile_cublas_enable_all();
	test_all();
	std::printf("-------\n");
}
