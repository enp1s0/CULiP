#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hipblas.h>
#include <CULiP/cublas.hpp>

struct op_gemm;
struct op_gemmEx;

template <class T, class Op>
hipblasStatus_t gemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##gemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_GEMM(H, half);
GEMM_OP_GEMM(S, float);
GEMM_OP_GEMM(D, double);
GEMM_OP_GEMM(C, hipComplex);
GEMM_OP_GEMM(Z, hipDoubleComplex);

// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
#define GEMM_OP_GEMM_EX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return hipblasGemmEx(handle, transa, transb, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_OP_GEMM_EX(HIP_R_16F, half);
GEMM_OP_GEMM_EX(HIP_R_32F, float);
GEMM_OP_GEMM_EX(HIP_R_64F, double);
GEMM_OP_GEMM_EX(HIP_C_32F, hipComplex);
GEMM_OP_GEMM_EX(HIP_C_64F, hipDoubleComplex);

// -------------
// GemmBatched
// -------------
template <class T, class Op>
hipblasStatus_t gemm_batched(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T **A, int lda,
                           const T **B, int ldb, const T *beta, T **C,
                           int ldc, int batchCount);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_BATCHED_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm_batched<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A[], int lda,\
                           const type *B[], int ldb, const type *beta, type *C[],\
                           int ldc, int batchCount) {\
	return cublas##short_type##gemmBatched(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, batchCount);\
}
GEMM_BATCHED_OP_GEMM(H, half);
GEMM_BATCHED_OP_GEMM(S, float);
GEMM_BATCHED_OP_GEMM(D, double);
GEMM_BATCHED_OP_GEMM(C, hipComplex);
GEMM_BATCHED_OP_GEMM(Z, hipDoubleComplex);
// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
#define GEMM_BATCHED_OP_GEMMEX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm_batched<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A[], int lda,\
                           const type *B[], int ldb, const type *beta, type *C[],\
                           int ldc, int batchCount) {\
	return hipblasGemmBatchedEx(handle, transa, transb, m, n, k, alpha, reinterpret_cast<const void**>(A), cuda_data_type, lda, reinterpret_cast<const void**>(B), cuda_data_type, ldb, beta, reinterpret_cast<void**>(C), cuda_data_type, ldc, batchCount, cuda_data_type, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_BATCHED_OP_GEMMEX(HIP_R_16F, half);
GEMM_BATCHED_OP_GEMMEX(HIP_R_32F, float);
GEMM_BATCHED_OP_GEMMEX(HIP_R_64F, double);
GEMM_BATCHED_OP_GEMMEX(HIP_C_32F, hipComplex);
GEMM_BATCHED_OP_GEMMEX(HIP_C_64F, hipDoubleComplex);

// -------------
// Gemv
// -------------
template <class T>
hipblasStatus_t gemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const T *alpha, const T *A, int lda,
                           const T *x, int incx, const T *beta, T *y,
                           int incy);
#define GEMM_OP_GEMV(short_type, type)\
template <>\
hipblasStatus_t gemv<type>(hipblasHandle_t handle, hipblasOperation_t trans,\
                           int m, int n,\
                           const type *alpha, const type *A, int lda,\
                           const type *x, int incx, const type *beta, type *y,\
                           int incy) {\
	return cublas##short_type##gemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);\
}
GEMM_OP_GEMV(S, float);
GEMM_OP_GEMV(D, double);
GEMM_OP_GEMV(C, hipComplex);
GEMM_OP_GEMV(Z, hipDoubleComplex);

// -------------
// Gbmv
// -------------
template <class T>
hipblasStatus_t gbmv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n, int kl, int ku,
                           const T *alpha, const T *A, int lda,
                           const T *x, int incx, const T *beta, T *y,
                           int incy);
#define GEMM_OP_GBMV(short_type, type)\
template <>\
hipblasStatus_t gbmv<type>(hipblasHandle_t handle, hipblasOperation_t trans,\
                           int m, int n, int kl, int ku,\
                           const type *alpha, const type *A, int lda,\
                           const type *x, int incx, const type *beta, type *y,\
                           int incy) {\
	return cublas##short_type##gbmv(handle, trans, m, n, kl, ku, alpha, A, lda, x, incx, beta, y, incy);\
}
GEMM_OP_GBMV(S, float);
GEMM_OP_GBMV(D, double);
GEMM_OP_GBMV(C, hipComplex);
GEMM_OP_GBMV(Z, hipDoubleComplex);

// -------------
// Syrk
// -------------
template <class T>
hipblasStatus_t syrk(hipblasHandle_t handle,
		                       hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYRK(short_type, type)\
template <>\
hipblasStatus_t syrk<type>(hipblasHandle_t handle, hipblasFillMode_t uplo,\
		                       hipblasOperation_t trans,\
                           int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##syrk(handle, uplo, trans, n, k, alpha, A, lda, beta, C, ldc);\
}
GEMM_OP_SYRK(S, float);
GEMM_OP_SYRK(D, double);
GEMM_OP_SYRK(C, hipComplex);
GEMM_OP_SYRK(Z, hipDoubleComplex);

// -------------
// Syr2k
// -------------
template <class T>
hipblasStatus_t syr2k(hipblasHandle_t handle,
		                       hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int n, int k,
                           const T *alpha,
													 const T *A, int lda,
													 const T *B, int ldb,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYR2K(short_type, type)\
template <>\
hipblasStatus_t syr2k<type>(hipblasHandle_t handle, hipblasFillMode_t uplo,\
		                       hipblasOperation_t trans,\
                           int n, int k,\
                           const type *alpha, \
													 const type *A, int lda,\
													 const type *B, int ldb,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##syr2k(handle, uplo, trans, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_SYR2K(S, float);
GEMM_OP_SYR2K(D, double);
GEMM_OP_SYR2K(C, hipComplex);
GEMM_OP_SYR2K(Z, hipDoubleComplex);

// -------------
// Symm
// -------------
template <class T>
hipblasStatus_t symm(hipblasHandle_t handle, hipblasSideMode_t size,
		                       hipblasFillMode_t uplo,
                           int m, int n,
                           const T *alpha,
													 const T *A, int lda,
													 const T *B, int ldb,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYMM(short_type, type)\
template <>\
hipblasStatus_t symm<type>(hipblasHandle_t handle, hipblasSideMode_t side,\
		                       hipblasFillMode_t uplo,\
                           int m, int n,\
                           const type *alpha, \
													 const type *A, int lda,\
													 const type *B, int ldb,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##symm(handle, side, uplo, m, n, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_SYMM(S, float);
GEMM_OP_SYMM(D, double);
GEMM_OP_SYMM(C, hipComplex);
GEMM_OP_SYMM(Z, hipDoubleComplex);

// -------------
// Gemm3m
// -------------
template <class T>
hipblasStatus_t gemm3m(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_OP_GEMM3M(short_type, type)\
template <>\
hipblasStatus_t gemm3m<type>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##gemm3m(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_GEMM3M(C, hipComplex);
GEMM_OP_GEMM3M(Z, hipDoubleComplex);


template <class T>
T convert(const double a) {return static_cast<T>(a);}
template <> hipComplex       convert<hipComplex      >(const double a) {return make_float2(a, 0);}
template <> hipDoubleComplex convert<hipDoubleComplex>(const double a) {return make_double2(a, 0);}

template <class T, class Op>
void gemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T, class Op>
void gemm_batched_test() {
	const int n = 1lu << 7;
	const int batch_size = 1u << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T** mat_a_array;
	T** mat_b_array;
	T** mat_c_array;

	hipHostMalloc(&mat_a_array, sizeof(T*) * batch_size);
	hipHostMalloc(&mat_b_array, sizeof(T*) * batch_size);
	hipHostMalloc(&mat_c_array, sizeof(T*) * batch_size);

	for (unsigned i = 0; i < batch_size; i++) {
		T* ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_a_array[i] = ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_b_array[i] = ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_c_array[i] = ptr;
	}

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm_batched<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			(const T**)mat_a_array, n,
			(const T**)mat_b_array, n,
			&beta,
			mat_c_array, n,
			batch_size
			);

	hipblasDestroy(cublas_handle);

	for (unsigned i = 0; i < batch_size; i++) {
		hipFree(mat_a_array[i]);
		hipFree(mat_b_array[i]);
		hipFree(mat_c_array[i]);
	}
	hipHostFree(mat_a_array);
	hipHostFree(mat_b_array);
	hipHostFree(mat_c_array);
}

template <class T>
void gemv_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemv<T>(
			cublas_handle,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			vec_x, 1,
			&beta,
			vec_y, 1
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void gbmv_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gbmv<T>(
			cublas_handle,
			HIPBLAS_OP_N,
			n, n, n / 10, n / 10,
			&alpha,
			mat_a, n,
			vec_x, 1,
			&beta,
			vec_y, 1
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void syrk_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	syrk<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_c);
}

template <class T>
void syr2k_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	syr2k<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void symm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	symm<T>(
			cublas_handle,
			HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void gemm3m_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm3m<T>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

void test_all() {
	gemm_test<double         , op_gemm  >();
	gemm_test<float          , op_gemm  >();
	gemm_test<half           , op_gemm  >();
	gemm_test<hipComplex      , op_gemm  >();
	gemm_test<hipDoubleComplex, op_gemm  >();
	gemm_test<double         , op_gemmEx>();
	gemm_test<float          , op_gemmEx>();
	gemm_test<half           , op_gemmEx>();
	gemm_test<hipComplex      , op_gemmEx>();
	gemm_test<hipDoubleComplex, op_gemmEx>();

	gemm_batched_test<double         , op_gemm  >();
	gemm_batched_test<float          , op_gemm  >();
	gemm_batched_test<half           , op_gemm  >();
	gemm_batched_test<hipComplex      , op_gemm  >();
	gemm_batched_test<hipDoubleComplex, op_gemm  >();
	gemm_batched_test<double         , op_gemmEx>();
	gemm_batched_test<float          , op_gemmEx>();
	gemm_batched_test<half           , op_gemmEx>();
	gemm_batched_test<hipComplex      , op_gemmEx>();
	gemm_batched_test<hipDoubleComplex, op_gemmEx>();

	gemv_test<double         >();
	gemv_test<float          >();
	gemv_test<hipComplex      >();
	gemv_test<hipDoubleComplex>();

	gbmv_test<double         >();
	gbmv_test<float          >();
	gbmv_test<hipComplex      >();
	gbmv_test<hipDoubleComplex>();

	syrk_test<double         >();
	syrk_test<float          >();
	syrk_test<hipComplex      >();
	syrk_test<hipDoubleComplex>();

	symm_test<double         >();
	symm_test<float          >();
	symm_test<hipComplex      >();
	symm_test<hipDoubleComplex>();

	syr2k_test<double         >();
	syr2k_test<float          >();
	syr2k_test<hipComplex      >();
	syr2k_test<hipDoubleComplex>();

	gemm3m_test<hipComplex      >();
	gemm3m_test<hipDoubleComplex>();
}

int main(){
	std::printf("Without profiling\n");
	CULiP_profile_cublas_disable_all();
	test_all();
	std::printf("-------\n");

	std::printf("With profiling\n");
	CULiP_profile_cublas_enable_all();
	test_all();
	std::printf("-------\n");
}
