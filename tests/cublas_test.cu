#include <iostream>
#include <hipblas.h>
#include <hipblas.h>

void sgemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = 1.f;
	const auto beta  = 0.f;

	float* mat_a;
	float* mat_b;
	float* mat_c;

	hipMalloc(&mat_a, sizeof(float) * n * n);
	hipMalloc(&mat_b, sizeof(float) * n * n);
	hipMalloc(&mat_c, sizeof(float) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	hipblasSgemm(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

int main(){
	sgemm_test();
}
