#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hipblas.h>
#include <CULiP/cublas.hpp>

template <class T>
struct get_real_type {
	using type = void;
};
template <> struct get_real_type<hipComplex> {using type = float;};
template <> struct get_real_type<hipDoubleComplex> {using type = double;};

struct op_gemm;
struct op_gemmEx;

template <class T, class Op>
hipblasStatus_t gemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##gemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_GEMM(H, half);
GEMM_OP_GEMM(S, float);
GEMM_OP_GEMM(D, double);
GEMM_OP_GEMM(C, hipComplex);
GEMM_OP_GEMM(Z, hipDoubleComplex);

// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
#define GEMM_OP_GEMM_EX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return hipblasGemmEx(handle, transa, transb, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_OP_GEMM_EX(HIP_R_16F, half);
GEMM_OP_GEMM_EX(HIP_R_32F, float);
GEMM_OP_GEMM_EX(HIP_R_64F, double);
GEMM_OP_GEMM_EX(HIP_C_32F, hipComplex);
GEMM_OP_GEMM_EX(HIP_C_64F, hipDoubleComplex);

// -------------
// GemmBatched
// -------------
template <class T, class Op>
hipblasStatus_t gemm_batched(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T **A, int lda,
                           const T **B, int ldb, const T *beta, T **C,
                           int ldc, int batchCount);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_BATCHED_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm_batched<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A[], int lda,\
                           const type *B[], int ldb, const type *beta, type *C[],\
                           int ldc, int batchCount) {\
	return cublas##short_type##gemmBatched(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, batchCount);\
}
GEMM_BATCHED_OP_GEMM(H, half);
GEMM_BATCHED_OP_GEMM(S, float);
GEMM_BATCHED_OP_GEMM(D, double);
GEMM_BATCHED_OP_GEMM(C, hipComplex);
GEMM_BATCHED_OP_GEMM(Z, hipDoubleComplex);
// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
#define GEMM_BATCHED_OP_GEMMEX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm_batched<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A[], int lda,\
                           const type *B[], int ldb, const type *beta, type *C[],\
                           int ldc, int batchCount) {\
	return hipblasGemmBatchedEx(handle, transa, transb, m, n, k, alpha, reinterpret_cast<const void**>(A), cuda_data_type, lda, reinterpret_cast<const void**>(B), cuda_data_type, ldb, beta, reinterpret_cast<void**>(C), cuda_data_type, ldc, batchCount, cuda_data_type, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_BATCHED_OP_GEMMEX(HIP_R_16F, half);
GEMM_BATCHED_OP_GEMMEX(HIP_R_32F, float);
GEMM_BATCHED_OP_GEMMEX(HIP_R_64F, double);
GEMM_BATCHED_OP_GEMMEX(HIP_C_32F, hipComplex);
GEMM_BATCHED_OP_GEMMEX(HIP_C_64F, hipDoubleComplex);

// -------------
// GemmStridedBatched
// -------------
template <class T, class Op>
hipblasStatus_t gemm_strided_batched(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda, long long int strideA,
                           const T *B, int ldb, long long int strideB, const T *beta, T *C,
                           int ldc, long long int strideC, int batchCount);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_STRIDED_BATCHED_OP_GEMM(short_type, type)\
template <>\
hipblasStatus_t gemm_strided_batched<type , op_gemm>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda, long long int strideA,\
                           const type *B, int ldb, long long int strideB, const type *beta, type *C,\
                           int ldc, long long int strideC, int batchCount) {\
	return cublas##short_type##gemmStridedBatched(handle, transa, transb, m, n, k, alpha, A, lda, strideA, B, ldb, strideB, beta, C, ldc, strideC, batchCount);\
}
GEMM_STRIDED_BATCHED_OP_GEMM(H, half);
GEMM_STRIDED_BATCHED_OP_GEMM(S, float);
GEMM_STRIDED_BATCHED_OP_GEMM(D, double);
GEMM_STRIDED_BATCHED_OP_GEMM(C, hipComplex);
GEMM_STRIDED_BATCHED_OP_GEMM(Z, hipDoubleComplex);
// -----------------------------------------------------
// op_gemmEx
// -----------------------------------------------------
#define GEMM_STRIDED_BATCHED_OP_GEMMEX(cuda_data_type, type)\
template <>\
hipblasStatus_t gemm_strided_batched<type , op_gemmEx>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda, long long int strideA,\
                           const type *B, int ldb, long long int strideB, const type *beta, type *C,\
                           int ldc, long long int strideC, int batchCount) {\
	return hipblasGemmStridedBatchedEx(handle, transa, transb, m, n, k, alpha, reinterpret_cast<const void*>(A), cuda_data_type, lda, strideA, reinterpret_cast<const void*>(B), cuda_data_type, ldb, strideB, beta, reinterpret_cast<void*>(C), cuda_data_type, ldc, strideC, batchCount, cuda_data_type, HIPBLAS_GEMM_DEFAULT);\
}
GEMM_STRIDED_BATCHED_OP_GEMMEX(HIP_R_16F, half);
GEMM_STRIDED_BATCHED_OP_GEMMEX(HIP_R_32F, float);
GEMM_STRIDED_BATCHED_OP_GEMMEX(HIP_R_64F, double);
GEMM_STRIDED_BATCHED_OP_GEMMEX(HIP_C_32F, hipComplex);
GEMM_STRIDED_BATCHED_OP_GEMMEX(HIP_C_64F, hipDoubleComplex);

// -------------
// Gemv
// -------------
template <class T>
hipblasStatus_t gemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const T *alpha, const T *A, int lda,
                           const T *x, int incx, const T *beta, T *y,
                           int incy);
#define GEMM_OP_GEMV(short_type, type)\
template <>\
hipblasStatus_t gemv<type>(hipblasHandle_t handle, hipblasOperation_t trans,\
                           int m, int n,\
                           const type *alpha, const type *A, int lda,\
                           const type *x, int incx, const type *beta, type *y,\
                           int incy) {\
	return cublas##short_type##gemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);\
}
GEMM_OP_GEMV(S, float);
GEMM_OP_GEMV(D, double);
GEMM_OP_GEMV(C, hipComplex);
GEMM_OP_GEMV(Z, hipDoubleComplex);

// -------------
// Gbmv
// -------------
template <class T>
hipblasStatus_t gbmv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n, int kl, int ku,
                           const T *alpha, const T *A, int lda,
                           const T *x, int incx, const T *beta, T *y,
                           int incy);
#define GEMM_OP_GBMV(short_type, type)\
template <>\
hipblasStatus_t gbmv<type>(hipblasHandle_t handle, hipblasOperation_t trans,\
                           int m, int n, int kl, int ku,\
                           const type *alpha, const type *A, int lda,\
                           const type *x, int incx, const type *beta, type *y,\
                           int incy) {\
	return cublas##short_type##gbmv(handle, trans, m, n, kl, ku, alpha, A, lda, x, incx, beta, y, incy);\
}
GEMM_OP_GBMV(S, float);
GEMM_OP_GBMV(D, double);
GEMM_OP_GBMV(C, hipComplex);
GEMM_OP_GBMV(Z, hipDoubleComplex);

// -------------
// Ger
// -------------
template <class T>
hipblasStatus_t ger(hipblasHandle_t handle,
                           int m, int n,
                           const T *alpha,
                           const T *x, int incx, const T *y,
                           int incy, T *A, int lda);
#define GEMM_OP_GER(short_type, type)\
template <>\
hipblasStatus_t ger<type>(hipblasHandle_t handle,\
                           int m, int n,\
                           const type *alpha, \
                           const type *x, int incx, const type *y,\
                           int incy, type* A, int lda) {\
	return cublas##short_type##ger(handle, m, n, alpha, x, incx, y, incy, A, lda);\
}
GEMM_OP_GER(S, float);
GEMM_OP_GER(D, double);

template <class T>
hipblasStatus_t gerc(hipblasHandle_t handle,
                           int m, int n,
                           const T *alpha,
                           const T *x, int incx, const T *y,
                           int incy, T *A, int lda);
#define GEMM_OP_GERC(short_type, type)\
template <>\
hipblasStatus_t gerc<type>(hipblasHandle_t handle,\
                           int m, int n,\
                           const type *alpha, \
                           const type *x, int incx, const type *y,\
                           int incy, type* A, int lda) {\
	return cublas##short_type##gerc(handle, m, n, alpha, x, incx, y, incy, A, lda);\
}
GEMM_OP_GERC(C, hipComplex);
GEMM_OP_GERC(Z, hipDoubleComplex);

template <class T>
hipblasStatus_t geru(hipblasHandle_t handle,
                           int m, int n,
                           const T *alpha,
                           const T *x, int incx, const T *y,
                           int incy, T *A, int lda);
#define GEMM_OP_GERU(short_type, type)\
template <>\
hipblasStatus_t geru<type>(hipblasHandle_t handle,\
                           int m, int n,\
                           const type *alpha, \
                           const type *x, int incx, const type *y,\
                           int incy, type* A, int lda) {\
	return cublas##short_type##geru(handle, m, n, alpha, x, incx, y, incy, A, lda);\
}
GEMM_OP_GERU(C, hipComplex);
GEMM_OP_GERU(Z, hipDoubleComplex);

// -------------
// Syrk
// -------------
template <class T>
hipblasStatus_t syrk(hipblasHandle_t handle,
		                       hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYRK(short_type, type)\
template <>\
hipblasStatus_t syrk<type>(hipblasHandle_t handle, hipblasFillMode_t uplo,\
		                       hipblasOperation_t trans,\
                           int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##syrk(handle, uplo, trans, n, k, alpha, A, lda, beta, C, ldc);\
}
GEMM_OP_SYRK(S, float);
GEMM_OP_SYRK(D, double);
GEMM_OP_SYRK(C, hipComplex);
GEMM_OP_SYRK(Z, hipDoubleComplex);

// -------------
// Syr2k
// -------------
template <class T>
hipblasStatus_t syr2k(hipblasHandle_t handle,
		                       hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int n, int k,
                           const T *alpha,
													 const T *A, int lda,
													 const T *B, int ldb,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYR2K(short_type, type)\
template <>\
hipblasStatus_t syr2k<type>(hipblasHandle_t handle, hipblasFillMode_t uplo,\
		                       hipblasOperation_t trans,\
                           int n, int k,\
                           const type *alpha, \
													 const type *A, int lda,\
													 const type *B, int ldb,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##syr2k(handle, uplo, trans, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_SYR2K(S, float);
GEMM_OP_SYR2K(D, double);
GEMM_OP_SYR2K(C, hipComplex);
GEMM_OP_SYR2K(Z, hipDoubleComplex);

// -------------
// Syrkx
// -------------
template <class T>
hipblasStatus_t syrkx(hipblasHandle_t handle,
		                       hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int n, int k,
                           const T *alpha,
													 const T *A, int lda,
													 const T *B, int ldb,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYRKX(short_type, type)\
template <>\
hipblasStatus_t syrkx<type>(hipblasHandle_t handle, hipblasFillMode_t uplo,\
		                       hipblasOperation_t trans,\
                           int n, int k,\
                           const type *alpha, \
													 const type *A, int lda,\
													 const type *B, int ldb,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##syrkx(handle, uplo, trans, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_SYRKX(S, float);
GEMM_OP_SYRKX(D, double);
GEMM_OP_SYRKX(C, hipComplex);
GEMM_OP_SYRKX(Z, hipDoubleComplex);

// -------------
// Symm
// -------------
template <class T>
hipblasStatus_t symm(hipblasHandle_t handle, hipblasSideMode_t size,
		                       hipblasFillMode_t uplo,
                           int m, int n,
                           const T *alpha,
													 const T *A, int lda,
													 const T *B, int ldb,
                           const T *beta , T *C, int ldc
                           );
#define GEMM_OP_SYMM(short_type, type)\
template <>\
hipblasStatus_t symm<type>(hipblasHandle_t handle, hipblasSideMode_t side,\
		                       hipblasFillMode_t uplo,\
                           int m, int n,\
                           const type *alpha, \
													 const type *A, int lda,\
													 const type *B, int ldb,\
                           const type *beta, type *C, int ldc\
                           ) {\
	return cublas##short_type##symm(handle, side, uplo, m, n, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_SYMM(S, float);
GEMM_OP_SYMM(D, double);
GEMM_OP_SYMM(C, hipComplex);
GEMM_OP_SYMM(Z, hipDoubleComplex);

// -------------
// Trmm
// -------------
template <class T>
hipblasStatus_t trmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const T *alpha,
													 const T *A, int lda,
													 const T *B, int ldb,
                           T *C, int ldc
                           );
#define GEMM_OP_TRMM(short_type, type)\
template <>\
hipblasStatus_t trmm<type>(hipblasHandle_t handle, \
                           hipblasSideMode_t side, hipblasFillMode_t uplo, \
                           hipblasOperation_t trans, hipblasDiagType_t diag, \
                           int m, int n,\
                           const type *alpha, \
													 const type *A, int lda,\
													 const type *B, int ldb,\
                           type *C, int ldc\
                           ) {\
	return cublas##short_type##trmm(handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb, C, ldc);\
}
GEMM_OP_TRMM(S, float);
GEMM_OP_TRMM(D, double);
GEMM_OP_TRMM(C, hipComplex);
GEMM_OP_TRMM(Z, hipDoubleComplex);

// -------------
// Trsm
// -------------
template <class T>
hipblasStatus_t trsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const T *alpha,
													 const T *A, int lda,
                           T *B, int ldb
                           );
#define GEMM_OP_TRSM(short_type, type)\
template <>\
hipblasStatus_t trsm<type>(hipblasHandle_t handle, \
                           hipblasSideMode_t side, hipblasFillMode_t uplo, \
                           hipblasOperation_t trans, hipblasDiagType_t diag, \
                           int m, int n,\
                           const type *alpha, \
													 const type *A, int lda,\
                           type *B, int ldb\
                           ) {\
	return cublas##short_type##trsm(handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb);\
}
GEMM_OP_TRSM(S, float);
GEMM_OP_TRSM(D, double);
GEMM_OP_TRSM(C, hipComplex);
GEMM_OP_TRSM(Z, hipDoubleComplex);

// -------------
// Trsm batched
// -------------
template <class T>
hipblasStatus_t trsm_batched(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const T *alpha,
													 const T *const A[], int lda,
                           T * const B[], int ldb,
													 int batchCount
                           );
#define GEMM_OP_TRSM_BATCHED(short_type, type)\
template <>\
hipblasStatus_t trsm_batched<type>(hipblasHandle_t handle, \
                           hipblasSideMode_t side, hipblasFillMode_t uplo, \
                           hipblasOperation_t trans, hipblasDiagType_t diag, \
                           int m, int n,\
                           const type *alpha, \
													 const type * const A[], int lda,\
                           type * const B[], int ldb,\
													 int batchCount\
                           ) {\
	return cublas##short_type##trsmBatched(handle, side, uplo, trans, diag, m, n, alpha, A, lda, B, ldb, batchCount);\
}
GEMM_OP_TRSM_BATCHED(S, float);
GEMM_OP_TRSM_BATCHED(D, double);
GEMM_OP_TRSM_BATCHED(C, hipComplex);
GEMM_OP_TRSM_BATCHED(Z, hipDoubleComplex);

// -----------------------------------------------------
// hemm
// -----------------------------------------------------
template <class T>
hipblasStatus_t hemm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           int m, int n,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
#define GEMM_OP_HEMM(short_type, type)\
template <>\
hipblasStatus_t hemm<type>(hipblasHandle_t handle,\
                           hipblasSideMode_t side, hipblasFillMode_t uplo, \
                           int m, int n, \
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##hemm(handle, side, uplo, m, n, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_HEMM(C, hipComplex);
GEMM_OP_HEMM(Z, hipDoubleComplex);

// -----------------------------------------------------
// herk
// -----------------------------------------------------
template <class T, class RT>
hipblasStatus_t herk(hipblasHandle_t handle,
                           hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int m, int n,
                           const RT *alpha, const T *A, int lda,
                           const RT *beta, T *C,
                           int ldc);
#define GEMM_OP_HERK(short_type, type, real_type)\
template <>\
hipblasStatus_t herk<type, real_type>(hipblasHandle_t handle,\
                           hipblasFillMode_t uplo, hipblasOperation_t trans, \
                           int m, int n, \
                           const real_type *alpha, const type *A, int lda,\
                           const real_type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##herk(handle, uplo, trans, m, n, alpha, A, lda, beta, C, ldc);\
}
GEMM_OP_HERK(C, hipComplex, float);
GEMM_OP_HERK(Z, hipDoubleComplex, double);

// -----------------------------------------------------
// her2k
// -----------------------------------------------------
template <class T, class RT>
hipblasStatus_t her2k(hipblasHandle_t handle,
                           hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int m, int n,
                           const T *alpha,
                           const T *A, int lda,
                           const T *B, int ldb,
                           const RT *beta, T *C,
                           int ldc);
#define GEMM_OP_HER2K(short_type, type, real_type)\
template <>\
hipblasStatus_t her2k<type, real_type>(hipblasHandle_t handle,\
                           hipblasFillMode_t uplo, hipblasOperation_t trans, \
                           int m, int n, \
                           const type *alpha,\
                           const type *A, int lda,\
                           const type *B, int ldb,\
                           const real_type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##her2k(handle, uplo, trans, m, n, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_HER2K(C, hipComplex, float);
GEMM_OP_HER2K(Z, hipDoubleComplex, double);

// -----------------------------------------------------
// herkx
// -----------------------------------------------------
template <class T, class RT>
hipblasStatus_t herkx(hipblasHandle_t handle,
                           hipblasFillMode_t uplo, hipblasOperation_t trans,
                           int m, int n,
                           const T *alpha,
                           const T *A, int lda,
                           const T *B, int ldb,
                           const RT *beta, T *C,
                           int ldc);
#define GEMM_OP_HERKX(short_type, type, real_type)\
template <>\
hipblasStatus_t herkx<type, real_type>(hipblasHandle_t handle,\
                           hipblasFillMode_t uplo, hipblasOperation_t trans, \
                           int m, int n, \
                           const type *alpha,\
                           const type *A, int lda,\
                           const type *B, int ldb,\
                           const real_type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##herkx(handle, uplo, trans, m, n, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_HERKX(C, hipComplex, float);
GEMM_OP_HERKX(Z, hipDoubleComplex, double);

// -------------
// Gemm3m
// -------------
template <class T>
hipblasStatus_t gemm3m(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const T *alpha, const T *A, int lda,
                           const T *B, int ldb, const T *beta, T *C,
                           int ldc);
// -----------------------------------------------------
// op_gemm
// -----------------------------------------------------
#define GEMM_OP_GEMM3M(short_type, type)\
template <>\
hipblasStatus_t gemm3m<type>(hipblasHandle_t handle, hipblasOperation_t transa,\
                           hipblasOperation_t transb, int m, int n, int k,\
                           const type *alpha, const type *A, int lda,\
                           const type *B, int ldb, const type *beta, type *C,\
                           int ldc) {\
	return cublas##short_type##gemm3m(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);\
}
GEMM_OP_GEMM3M(C, hipComplex);
GEMM_OP_GEMM3M(Z, hipDoubleComplex);


template <class T>
T convert(const double a) {return static_cast<T>(a);}
template <> hipComplex       convert<hipComplex      >(const double a) {return make_float2(a, 0);}
template <> hipDoubleComplex convert<hipDoubleComplex>(const double a) {return make_double2(a, 0);}

template <class T, class Op>
void gemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T, class Op>
void gemm_batched_test() {
	const int n = 1lu << 7;
	const int batch_size = 1u << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T** mat_a_array;
	T** mat_b_array;
	T** mat_c_array;

	hipHostMalloc(&mat_a_array, sizeof(T*) * batch_size);
	hipHostMalloc(&mat_b_array, sizeof(T*) * batch_size);
	hipHostMalloc(&mat_c_array, sizeof(T*) * batch_size);

	for (unsigned i = 0; i < batch_size; i++) {
		T* ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_a_array[i] = ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_b_array[i] = ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_c_array[i] = ptr;
	}

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm_batched<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			(const T**)mat_a_array, n,
			(const T**)mat_b_array, n,
			&beta,
			mat_c_array, n,
			batch_size
			);

	hipblasDestroy(cublas_handle);

	for (unsigned i = 0; i < batch_size; i++) {
		hipFree(mat_a_array[i]);
		hipFree(mat_b_array[i]);
		hipFree(mat_c_array[i]);
	}
	hipHostFree(mat_a_array);
	hipHostFree(mat_b_array);
	hipHostFree(mat_c_array);
}

template <class T, class Op>
void gemm_strided_batched_test() {
	const int n = 1lu << 7;
	const int batch_size = 1u << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipHostMalloc(&mat_a, sizeof(T) * n * n * batch_size);
	hipHostMalloc(&mat_b, sizeof(T) * n * n * batch_size);
	hipHostMalloc(&mat_c, sizeof(T) * n * n * batch_size);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm_strided_batched<T, Op>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n, n * n,
			mat_b, n, n * n,
			&beta,
			mat_c, n, n * n,
			batch_size
			);

	hipblasDestroy(cublas_handle);
	hipHostFree(mat_a);
	hipHostFree(mat_b);
	hipHostFree(mat_c);
}

template <class T>
void gemv_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemv<T>(
			cublas_handle,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			vec_x, 1,
			&beta,
			vec_y, 1
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void gbmv_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gbmv<T>(
			cublas_handle,
			HIPBLAS_OP_N,
			n, n, n / 10, n / 10,
			&alpha,
			mat_a, n,
			vec_x, 1,
			&beta,
			vec_y, 1
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void ger_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	ger<T>(
			cublas_handle,
			n, n,
			&alpha,
			vec_x, 1,
			vec_y, 1,
			mat_a, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void gerc_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gerc<T>(
			cublas_handle,
			n, n,
			&alpha,
			vec_x, 1,
			vec_y, 1,
			mat_a, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void geru_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);

	T* mat_a;
	T* vec_x;
	T* vec_y;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&vec_x, sizeof(T) * n);
	hipMalloc(&vec_y, sizeof(T) * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	geru<T>(
			cublas_handle,
			n, n,
			&alpha,
			vec_x, 1,
			vec_y, 1,
			mat_a, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(vec_x);
	hipFree(vec_y);
}

template <class T>
void syrk_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	syrk<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_c);
}

template <class T>
void syr2k_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	syr2k<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void syrkx_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	syrkx<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void symm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	symm<T>(
			cublas_handle,
			HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void trmm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	trmm<T>(
			cublas_handle,
			HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void trsm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);

	T* mat_a;
	T* mat_b;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	trsm<T>(
			cublas_handle,
			HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
}

template <class T>
void trsm_batched_test() {
	const std::size_t n = 1lu << 8;
	const std::size_t batch_size = 1lu << 5;
	const auto alpha = convert<T>(1);

	T** mat_a_array;
	T** mat_b_array;

	hipHostMalloc(&mat_a_array, sizeof(T*) * batch_size);
	hipHostMalloc(&mat_b_array, sizeof(T*) * batch_size);

	for (unsigned i = 0; i < batch_size; i++) {
		T* ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_a_array[i] = ptr;
		hipMalloc(&ptr, sizeof(T) * n * n);
		mat_b_array[i] = ptr;
	}

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	trsm_batched<T>(
			cublas_handle,
			HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
			n, n,
			&alpha,
			mat_a_array, n,
			mat_b_array, n,
			batch_size
			);

	hipblasDestroy(cublas_handle);

	for (unsigned i = 0; i < batch_size; i++) {
		hipFree(mat_a_array[i]);
		hipFree(mat_b_array[i]);
	}
	hipHostFree(mat_a_array);
	hipHostFree(mat_b_array);
}

template <class T>
void hemm_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	hemm<T>(
			cublas_handle,
			HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void herk_test() {
	using real_type = typename get_real_type<T>::type;
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<real_type>(1);
	const auto beta = convert<real_type>(0);

	T* mat_a;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	herk<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_c);
}

template <class T>
void her2k_test() {
	using real_type = typename get_real_type<T>::type;
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta = convert<real_type>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	her2k<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void herkx_test() {
	using real_type = typename get_real_type<T>::type;
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta = convert<real_type>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	herkx<T>(
			cublas_handle,
			HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
			n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

template <class T>
void gemm3m_test() {
	const std::size_t n = 1lu << 10;
	const auto alpha = convert<T>(1);
	const auto beta  = convert<T>(0);

	T* mat_a;
	T* mat_b;
	T* mat_c;

	hipMalloc(&mat_a, sizeof(T) * n * n);
	hipMalloc(&mat_b, sizeof(T) * n * n);
	hipMalloc(&mat_c, sizeof(T) * n * n);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	gemm3m<T>(
			cublas_handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			n, n, n,
			&alpha,
			mat_a, n,
			mat_b, n,
			&beta,
			mat_c, n
			);

	hipblasDestroy(cublas_handle);
	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

void test_all() {
	gemv_test<double         >();
	gemv_test<float          >();
	gemv_test<hipComplex      >();
	gemv_test<hipDoubleComplex>();

	gbmv_test<double         >();
	gbmv_test<float          >();
	gbmv_test<hipComplex      >();
	gbmv_test<hipDoubleComplex>();

	ger_test<double         >();
	ger_test<float          >();
	gerc_test<hipComplex      >();
	gerc_test<hipDoubleComplex>();
	geru_test<hipComplex      >();
	geru_test<hipDoubleComplex>();

	gemm_test<double         , op_gemm  >();
	gemm_test<float          , op_gemm  >();
	gemm_test<half           , op_gemm  >();
	gemm_test<hipComplex      , op_gemm  >();
	gemm_test<hipDoubleComplex, op_gemm  >();
	gemm_test<double         , op_gemmEx>();
	gemm_test<float          , op_gemmEx>();
	gemm_test<half           , op_gemmEx>();
	gemm_test<hipComplex      , op_gemmEx>();
	gemm_test<hipDoubleComplex, op_gemmEx>();

	gemm_batched_test<double         , op_gemm  >();
	gemm_batched_test<float          , op_gemm  >();
	gemm_batched_test<half           , op_gemm  >();
	gemm_batched_test<hipComplex      , op_gemm  >();
	gemm_batched_test<hipDoubleComplex, op_gemm  >();
	gemm_batched_test<double         , op_gemmEx>();
	gemm_batched_test<float          , op_gemmEx>();
	gemm_batched_test<half           , op_gemmEx>();
	gemm_batched_test<hipComplex      , op_gemmEx>();
	gemm_batched_test<hipDoubleComplex, op_gemmEx>();

	gemm_strided_batched_test<double         , op_gemm  >();
	gemm_strided_batched_test<float          , op_gemm  >();
	gemm_strided_batched_test<half           , op_gemm  >();
	gemm_strided_batched_test<hipComplex      , op_gemm  >();
	gemm_strided_batched_test<hipDoubleComplex, op_gemm  >();
	gemm_strided_batched_test<double         , op_gemmEx>();
	gemm_strided_batched_test<float          , op_gemmEx>();
	gemm_strided_batched_test<half           , op_gemmEx>();
	gemm_strided_batched_test<hipComplex      , op_gemmEx>();
	gemm_strided_batched_test<hipDoubleComplex, op_gemmEx>();

	syrk_test<double         >();
	syrk_test<float          >();
	syrk_test<hipComplex      >();
	syrk_test<hipDoubleComplex>();

	symm_test<double         >();
	symm_test<float          >();
	symm_test<hipComplex      >();
	symm_test<hipDoubleComplex>();

	syr2k_test<double         >();
	syr2k_test<float          >();
	syr2k_test<hipComplex      >();
	syr2k_test<hipDoubleComplex>();

	syrkx_test<double         >();
	syrkx_test<float          >();
	syrkx_test<hipComplex      >();
	syrkx_test<hipDoubleComplex>();

	trmm_test<double         >();
	trmm_test<float          >();
	trmm_test<hipComplex      >();
	trmm_test<hipDoubleComplex>();

	trsm_test<double         >();
	trsm_test<float          >();
	trsm_test<hipComplex      >();
	trsm_test<hipDoubleComplex>();

	trsm_batched_test<double         >();
	trsm_batched_test<float          >();
	trsm_batched_test<hipComplex      >();
	trsm_batched_test<hipDoubleComplex>();

	hemm_test<hipComplex      >();
	hemm_test<hipDoubleComplex>();

	herk_test<hipComplex      >();
	herk_test<hipDoubleComplex>();

	her2k_test<hipComplex      >();
	her2k_test<hipDoubleComplex>();

	herkx_test<hipComplex      >();
	herkx_test<hipDoubleComplex>();

	gemm3m_test<hipComplex      >();
	gemm3m_test<hipDoubleComplex>();
}

int main(){
	std::printf("Without profiling\n");
	CULiP_profile_cublas_disable_all();
	test_all();
	std::printf("-------\n");

	std::printf("With profiling\n");
	CULiP_profile_cublas_enable_all();
	test_all();
	std::printf("-------\n");
}
