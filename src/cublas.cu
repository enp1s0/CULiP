#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <CULiP/cublas.hpp>
#include <cu_exp_statistics.hpp>
#include <cu_cutoff.hpp>
#include "utils.hpp"

#define CULIP_CUBLAS_LIBRARY_NAME       "libcublas.so"
#define CULIP_CUBLAS_ENV_NAME           "CULIP_CUBLAS_LIB_PATH"
#define CULIP_CUBLAS_DISABLE_ENV_NAME   "CULIP_DISABLE_CUBLAS_PROFILING"
#define CULIP_EXP_STATS_ENABLE_ENV_NAME "CULIP_ENABLE_EXP_STATS"
#define CULIP_CUTOFF_THRESHOLD_ENV_NAME "CULIP_CUTOFF_THRESHOLD"

namespace {
mtk::cu_exp_statistics::result_t exp_stats(
		const void* const ptr,
		const std::size_t offset,
		const std::size_t m,
		const std::size_t n,
		const std::size_t ld,
		const hipStream_t hip_stream,
		hipDataType data_t
		) {
	mtk::cu_exp_statistics::result_t result;
	switch (data_t) {
	case HIP_R_64F:
		result = mtk::cu_exp_statistics::take_matrix_statistics(reinterpret_cast<const double*>(ptr) + offset, m, n, ld, hip_stream);
		break;
	case HIP_R_32F:
		result = mtk::cu_exp_statistics::take_matrix_statistics(reinterpret_cast<const float*>(ptr) + offset, m, n, ld, hip_stream);
		break;
	case HIP_R_16F:
		result = mtk::cu_exp_statistics::take_matrix_statistics(reinterpret_cast<const half*>(ptr) + offset, m, n, ld, hip_stream);
		break;
	case HIP_C_64F:
		result = mtk::cu_exp_statistics::take_matrix_statistics(reinterpret_cast<const double2*>(ptr) + offset, m, n, ld, hip_stream);
		break;
	case HIP_C_32F:
		result = mtk::cu_exp_statistics::take_matrix_statistics(reinterpret_cast<const float2*>(ptr) + offset, m, n, ld, hip_stream);
		break;
	case HIP_C_16F:
		result = mtk::cu_exp_statistics::take_matrix_statistics(reinterpret_cast<const half2*>(ptr) + offset, m, n, ld, hip_stream);
		break;
	default:
		break;
	}
	return result;
}

void cutoff(
		void* const ptr,
		const std::size_t offset,
		const std::size_t m,
		const std::size_t n,
		const std::size_t ld,
		double threshold,
		const hipStream_t hip_stream,
		hipDataType data_t
		) {
	switch (data_t) {
	case HIP_R_64F:
		mtk::cu_cutoff::cutoff_small_abs_values(reinterpret_cast<double*>(ptr) + offset, m, n, ld, threshold, hip_stream);
		break;
	case HIP_R_32F:
		mtk::cu_cutoff::cutoff_small_abs_values(reinterpret_cast<float*>(ptr) + offset, m, n, ld, threshold, hip_stream);
		break;
	case HIP_R_16F:
		mtk::cu_cutoff::cutoff_small_abs_values(reinterpret_cast<half*>(ptr) + offset, m, n, ld, threshold, hip_stream);
		break;
	case HIP_C_64F:
		mtk::cu_cutoff::cutoff_small_abs_values(reinterpret_cast<double2*>(ptr) + offset, m, n, ld, threshold, hip_stream);
		break;
	case HIP_C_32F:
		mtk::cu_cutoff::cutoff_small_abs_values(reinterpret_cast<float2*>(ptr) + offset, m, n, ld, threshold, hip_stream);
		break;
	case HIP_C_16F:
		mtk::cu_cutoff::cutoff_small_abs_values(reinterpret_cast<half2*>(ptr) + offset, m, n, ld, threshold, hip_stream);
		break;
	default:
		break;
	}
}
} // unnamed namespace

extern "C" {
// dlopen cache
void* CULiP_cublas_lib_handle_cache = NULL;

// Control profiling
// 0         = Profiling
// Otherwise = Not profiling
int CULiP_profiling_control_array[CULiP_cublas_enum_length] = {0};

// Controler setter
void CULiP_enable_cublas_profiling(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 0;
}
void CULiP_disable_cublas_profiling(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 1;
}
void CULiP_enable_cublas_all_profiling() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 0;
	}
}
void CULiP_disable_cublas_all_profiling() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 1;
	}
}

// hipDataType yo string
#define CULiP_CUBLAS_COMPUTE_T_CASE_STRING(compute_type) case compute_type: return #compute_type
extern "C" const char* CULiP_get_cublasComputeType_t_string(const hipblasComputeType_t compute_type) {
	switch(compute_type) {
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_16F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_16F_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_16F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_TF32);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32I);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32I_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_64F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_64F_PEDANTIC);
	default:
		break;
	}
	switch((hipDataType)compute_type) {
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_16F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_32F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_32I );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_64F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_8I  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_8U  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_16F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_32F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_32I );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_64F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_8I  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_8U  );
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasOperation_t_string(const hipblasOperation_t op) {
	switch(op) {
	case HIPBLAS_OP_N:
		return "N";
	case HIPBLAS_OP_T:
		return "T";
	case HIPBLAS_OP_C:
		return "C";
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasFillMode_t_string(const hipblasFillMode_t mode) {
	switch(mode) {
	case HIPBLAS_FILL_MODE_FULL:
		return "FULL";
	case HIPBLAS_FILL_MODE_LOWER:
		return "LOWER";
	case HIPBLAS_FILL_MODE_UPPER:
		return "UPPER";
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasSideMode_t_string(const hipblasSideMode_t mode) {
	switch(mode) {
	case HIPBLAS_SIDE_LEFT:
		return "LEFT";
	case HIPBLAS_SIDE_RIGHT:
		return "RIGHT";
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasDiagType_t_string(const hipblasDiagType_t type) {
	switch(type) {
	case HIPBLAS_DIAG_NON_UNIT:
		return "NON_UNIT";
	case HIPBLAS_DIAG_UNIT:
		return "UNIT";
	default:
		return "Unknown";
	}
}

// -------------------------------------------------
// GEMM
// -------------------------------------------------

// SGEMM
#define CULIP_FUNC_NAME hipblasSgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemm
#define CULIP_TYPE float
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMM
#define CULIP_FUNC_NAME hipblasDgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemm
#define CULIP_TYPE double
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// HGEMM
#define CULIP_FUNC_NAME hipblasHgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasHgemm
#define CULIP_TYPE half
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMM
#define CULIP_FUNC_NAME hipblasCgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemm
#define CULIP_TYPE hipComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME hipblasZgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, const void *B,
                            hipDataType Btype, int ldb, const void *beta,
                            void *C, hipDataType Ctype, int ldc,
                            hipblasComputeType_t computeType,
                            hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const int cutoff_flag = (CULiP_profiling_control_array[CULiP_cublasGemmEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUTOFF_THRESHOLD_ENV_NAME, false);
	if (cutoff_flag) {
		double threshold;
		try {
			const auto env_str = getenv(CULIP_CUTOFF_THRESHOLD_ENV_NAME);
			threshold	= std::stod(env_str);

			hipStream_t hip_stream;
			hipblasGetStream(handle, &hip_stream);
			cutoff(const_cast<void*>(A), 0, (transa == HIPBLAS_OP_N ? m : k), (transa == HIPBLAS_OP_N ? k : m), lda, threshold, hip_stream, Atype);
			cutoff(const_cast<void*>(B), 0, (transb == HIPBLAS_OP_N ? k : n), (transb == HIPBLAS_OP_N ? n : k), ldb, threshold, hip_stream, Btype);
		} catch(const std::exception& e) {
			CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Warning] invalid threshold (%s)\n", env_str));
		}
	}

	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-%s-%s-%s-%s-m%d-n%d-k%d", __func__,
				CULiP_get_cublasOperation_t_string(transa), CULiP_get_cublasOperation_t_string(transb),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Atype),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Btype),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Ctype),
				CULiP_get_cublasComputeType_t_string(computeType), m, n , k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	const int exp_stats_flag = (CULiP_profiling_control_array[CULiP_cublasGemmEx] == 0) && CULiP_is_profiling_enabled(CULIP_EXP_STATS_ENABLE_ENV_NAME, false);
	if (exp_stats_flag) {
		hipStream_t hip_stream;
		hipblasGetStream(handle, &hip_stream);
		CULiP_exp_stats a_stats;
		CULiP_exp_stats b_stats;
		snprintf(a_stats.name, a_stats.name_length - 1, "A");
		snprintf(b_stats.name, b_stats.name_length - 1, "B");
		a_stats.stats = exp_stats(A, 0, (transa == HIPBLAS_OP_N ? m : k), (transa == HIPBLAS_OP_N ? k : m), lda, hip_stream, Atype);
		b_stats.stats = exp_stats(B, 0, (transb == HIPBLAS_OP_N ? k : n), (transb == HIPBLAS_OP_N ? n : k), ldb, hip_stream, Btype);
		mtk::cu_exp_statistics::to_json(a_stats.stats);
		mtk::cu_exp_statistics::to_json(b_stats.stats);
		CULiP_launch_function(hip_stream, &CULiP_print_exp_stats_result, (void*)&a_stats);
		CULiP_launch_function(hip_stream, &CULiP_print_exp_stats_result, (void*)&b_stats);
	}

	return result;
#endif
}

// -------------------------------------------------
// GEMM_BATCHED
// -------------------------------------------------

// SGEMM
#define CULIP_FUNC_NAME hipblasSgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemmBatched
#define CULIP_TYPE float
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMM
#define CULIP_FUNC_NAME hipblasDgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemmBatched
#define CULIP_TYPE double
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// HGEMM
#define CULIP_FUNC_NAME hipblasHgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasHgemmBatched
#define CULIP_TYPE half
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMM
#define CULIP_FUNC_NAME hipblasCgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemmBatched
#define CULIP_TYPE hipComplex
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME hipblasZgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemmBatched
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

hipblasStatus_t hipblasGemmBatchedEx(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const void *alpha,
		const void *const Aarray[],
		hipDataType Atype,
		int lda,
		const void *const Barray[],
		hipDataType Btype,
		int ldb,
		const void *beta,
		void *const Carray[],
		hipDataType Ctype,
		int ldc,
		int batchCount,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmBatchedEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void* const[], hipDataType, int, const void* const[], hipDataType, int, const void*, void* const[], hipDataType, int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-%s-%s-%s-%s-m%d-n%d-k%d-batchCount%d", __func__,
				CULiP_get_cublasOperation_t_string(transa), CULiP_get_cublasOperation_t_string(transb),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Atype),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Btype),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Ctype),
				CULiP_get_cublasComputeType_t_string(computeType), m, n , k, batchCount);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, Aarray, Atype, lda, Barray, Btype, ldb, beta, Carray, Ctype, ldc, batchCount, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
#endif
}

// -------------------------------------------------
// GEMM_STRIDED_BATCHED
// -------------------------------------------------

// SGEMM
#define CULIP_FUNC_NAME hipblasSgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemmStridedBatched
#define CULIP_TYPE float
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMM
#define CULIP_FUNC_NAME hipblasDgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemmStridedBatched
#define CULIP_TYPE double
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// HGEMM
#define CULIP_FUNC_NAME hipblasHgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasHgemmStridedBatched
#define CULIP_TYPE half
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMM
#define CULIP_FUNC_NAME hipblasCgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemmStridedBatched
#define CULIP_TYPE hipComplex
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME hipblasZgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemmBatched
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const void *alpha,
		const void *A,
		hipDataType Atype,
		int lda,
		long long int strideA,
		const void *const B,
		hipDataType Btype,
		int ldb,
		long long int strideB,
		const void *beta,
		void *const C,
		hipDataType Ctype,
		int ldc,
		long long int strideC,
		int batchCount,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const int cutoff_flag = (CULiP_profiling_control_array[CULiP_cublasGemmStridedBatchedEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUTOFF_THRESHOLD_ENV_NAME, false);
	if (cutoff_flag) {
		double threshold;
		try {
			const auto env_str = getenv(CULIP_CUTOFF_THRESHOLD_ENV_NAME);
			threshold	= std::stod(env_str);

			hipStream_t hip_stream;
			hipblasGetStream(handle, &hip_stream);
			for (std::uint32_t i = 0; i < batchCount; i++) {
				cutoff(const_cast<void*>(A), i * strideA, (transa == HIPBLAS_OP_N ? m : k), (transa == HIPBLAS_OP_N ? k : m), lda, threshold, hip_stream, Atype);
				cutoff(const_cast<void*>(B), i * strideB, (transb == HIPBLAS_OP_N ? k : n), (transb == HIPBLAS_OP_N ? n : k), ldb, threshold, hip_stream, Btype);
			}
		} catch(const std::exception& e) {
			CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Warning] invalid threshold (%s)\n", env_str));
		}
	}

	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmStridedBatchedEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-%s-%s-%s-%s-m%d-n%d-k%d-batchCount%d", __func__,
				CULiP_get_cublasOperation_t_string(transa), CULiP_get_cublasOperation_t_string(transb),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Atype),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Btype),
				CULiP_get_cublasComputeType_t_string((hipblasComputeType_t)Ctype),
				CULiP_get_cublasComputeType_t_string(computeType), m, n , k, batchCount);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batchCount, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	const int exp_stats_flag = (CULiP_profiling_control_array[CULiP_cublasGemmStridedBatchedEx] == 0) && CULiP_is_profiling_enabled(CULIP_EXP_STATS_ENABLE_ENV_NAME, false);
	if (exp_stats_flag) {
		hipStream_t hip_stream;
		hipblasGetStream(handle, &hip_stream);
		CULiP_exp_stats a_stats;
		CULiP_exp_stats b_stats;
		snprintf(a_stats.name, a_stats.name_length - 1, "A");
		snprintf(b_stats.name, b_stats.name_length - 1, "B");
		for (std::uint32_t i = 0; i < batchCount; i++) {
			a_stats.stats += exp_stats(A, i * strideA, (transa == HIPBLAS_OP_N ? m : k), (transa == HIPBLAS_OP_N ? k : m), lda, hip_stream, Atype);
			b_stats.stats += exp_stats(B, i * strideB, (transb == HIPBLAS_OP_N ? k : n), (transb == HIPBLAS_OP_N ? n : k), ldb, hip_stream, Btype);
		}
		mtk::cu_exp_statistics::to_json(a_stats.stats);
		mtk::cu_exp_statistics::to_json(b_stats.stats);
		CULiP_launch_function(hip_stream, &CULiP_print_exp_stats_result, (void*)&a_stats);
		CULiP_launch_function(hip_stream, &CULiP_print_exp_stats_result, (void*)&b_stats);
	}

	return result;
#endif
}

// -------------------------------------------------
// GEMV
// -------------------------------------------------

// SGEMV
#define CULIP_FUNC_NAME hipblasSgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemv
#define CULIP_TYPE float
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMV
#define CULIP_FUNC_NAME hipblasDgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemv
#define CULIP_TYPE double
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMV
#define CULIP_FUNC_NAME hipblasCgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemv
#define CULIP_TYPE hipComplex
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMV
#define CULIP_FUNC_NAME hipblasZgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemv
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// GBMV
// -------------------------------------------------

// SGBMV
#define CULIP_FUNC_NAME hipblasSgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgbmv
#define CULIP_TYPE float
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGBMV
#define CULIP_FUNC_NAME hipblasDgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgbmv
#define CULIP_TYPE double
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGBMV
#define CULIP_FUNC_NAME hipblasCgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgbmv
#define CULIP_TYPE hipComplex
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGBMV
#define CULIP_FUNC_NAME hipblasZgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgbmv
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// GER
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSger
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSger
#define CULIP_TYPE float
#include "cublas.ger.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDger
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDger
#define CULIP_TYPE double
#include "cublas.ger.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCgerc
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgerc
#define CULIP_TYPE hipComplex
#include "cublas.ger.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCgeru
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgeru
#define CULIP_TYPE hipComplex
#include "cublas.ger.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZgerc
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgerc
#define CULIP_TYPE hipDoubleComplex
#include "cublas.ger.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZgeru
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgeru
#define CULIP_TYPE hipDoubleComplex
#include "cublas.ger.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SBMV
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsbmv
#define CULIP_TYPE float
#include "cublas.sbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsbmv
#define CULIP_TYPE double
#include "cublas.sbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SPMV
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSspmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSspmv
#define CULIP_TYPE float
#include "cublas.spmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDspmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDspmv
#define CULIP_TYPE double
#include "cublas.spmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYRK
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsyrk
#define CULIP_TYPE float
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsyrk
#define CULIP_TYPE double
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsyrk
#define CULIP_TYPE hipComplex
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsyrk
#define CULIP_TYPE hipDoubleComplex
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYR2K
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsyr2k
#define CULIP_TYPE float
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsyr2k
#define CULIP_TYPE double
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsyr2k
#define CULIP_TYPE hipComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsyr2k
#define CULIP_TYPE hipDoubleComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYRKX
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsyrkx
#define CULIP_TYPE float
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsyrkx
#define CULIP_TYPE double
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsyrkx
#define CULIP_TYPE hipComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsyrkx
#define CULIP_TYPE hipDoubleComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// GEMM3M
// -------------------------------------------------
// CGEMM
#define CULIP_FUNC_NAME cublasCgemm3m
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemm3m
#define CULIP_TYPE hipComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME cublasZgemm3m
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemm3m
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYMM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsymm
#define CULIP_TYPE float
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsymm
#define CULIP_TYPE double
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsymm
#define CULIP_TYPE hipComplex
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsymm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// TRMM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasStrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasStrmm
#define CULIP_TYPE float
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDtrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDtrmm
#define CULIP_TYPE double
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCtrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCtrmm
#define CULIP_TYPE hipComplex
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZtrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZtrmm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// TRSM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasStrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasStrsm
#define CULIP_TYPE float
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDtrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDtrsm
#define CULIP_TYPE double
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCtrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCtrsm
#define CULIP_TYPE hipComplex
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZtrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZtrsm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// TRSM Batched
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasStrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasStrsmBatched
#define CULIP_TYPE float
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDtrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDtrsmBatched
#define CULIP_TYPE double
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCtrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCtrsmBatched
#define CULIP_TYPE hipComplex
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZtrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZtrsmBatched
#define CULIP_TYPE hipDoubleComplex
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// HERK
// -------------------------------------------------
#define CULIP_FUNC_NAME hipblasCherk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCherk
#define CULIP_TYPE hipComplex
#define CULIP_REAL_TYPE float
#include "cublas.herk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
#undef CULIP_REAL_TYPE

#define CULIP_FUNC_NAME hipblasZherk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZherk
#define CULIP_TYPE hipDoubleComplex
#define CULIP_REAL_TYPE double
#include "cublas.herk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
#undef CULIP_REAL_TYPE

// -------------------------------------------------
// HEMM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasChemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasChemm
#define CULIP_TYPE hipComplex
#include "cublas.hemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZhemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZhemm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.hemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// HER2K
// -------------------------------------------------
#define CULIP_FUNC_NAME hipblasCher2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCher2k
#define CULIP_TYPE hipComplex
#define CULIP_REAL_TYPE float
#include "cublas.her2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
#undef CULIP_REAL_TYPE

#define CULIP_FUNC_NAME hipblasZher2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZher2k
#define CULIP_TYPE hipDoubleComplex
#define CULIP_REAL_TYPE double
#include "cublas.her2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
#undef CULIP_REAL_TYPE

// -------------------------------------------------
// HERKX
// -------------------------------------------------
#define CULIP_FUNC_NAME hipblasCherkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCherkx
#define CULIP_TYPE hipComplex
#define CULIP_REAL_TYPE float
#include "cublas.her2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
#undef CULIP_REAL_TYPE

#define CULIP_FUNC_NAME hipblasZherkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZherkx
#define CULIP_TYPE hipDoubleComplex
#define CULIP_REAL_TYPE double
#include "cublas.her2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
#undef CULIP_REAL_TYPE

} // extern "C"
