#include <hipblas.h>
#include <hipblas.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <CULiP/cublas.hpp>
#include "utils.hpp"

#define CULIP_CUBLAS_LIBRARY_NAME       "libcublas.so"
#define CULIP_CUBLAS_ENV_NAME           "CULIP_CUBLAS_LIB_PATH"
#define CULIP_CUBLAS_DISABLE_ENV_NAME   "CULIP_PROFILING_CUBLAS_DISABLE"

extern "C" {
// dlopen cache
void* CULiP_cublas_lib_handle_cache = NULL;

// Control profiling
// 0         = Profiling
// Otherwise = Not profiling
int CULiP_profiling_control_array[CULiP_cublas_enum_length] = {0};

// Controler setter
void CULiP_profile_cublas_enable(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 0;
}
void CULiP_profile_cublas_disable(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 1;
}
void CULiP_profile_cublas_enable_all() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 0;
	}
}
void CULiP_profile_cublas_disable_all() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 1;
	}
}

// hipDataType yo string
#define CULiP_CUBLAS_COMPUTE_T_CASE_STRING(compute_type) case compute_type: return #compute_type
extern "C" const char* CULiP_get_cublasComputeType_t_string(const hipblasComputeType_t compute_type) {
	switch(compute_type) {
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_16F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_16F_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_16F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_TF32);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32I);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32I_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_64F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_64F_PEDANTIC);
	default:
		break;
	}
	switch((hipDataType)compute_type) {
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_16F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_32F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_32I );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_64F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_8I  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_8U  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_16F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_32F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_32I );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_64F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_8I  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_8U  );
	default:
		return "Unknown";
	}
}

// -------------------------------------------------
// cuBLAS functions
// -------------------------------------------------

hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasSgemm] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const float*, const float*, int, const float*, int, const float*, float*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasDgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const double *alpha, const double *A, int lda,
                           const double *B, int ldb, const double *beta, double *C,
                           int ldc) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasDgemm] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const double*, const double*, int, const double*, int, const double*, double*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasHgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const half *alpha, const half *A, int lda,
                           const half *B, int ldb, const half *beta, half *C,
                           int ldc) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasHgemm] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const half*, const half*, int, const half*, int, const half*, half*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasCgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const hipComplex *alpha, const hipComplex *A, int lda,
                           const hipComplex *B, int ldb, const hipComplex *beta, hipComplex *C,
                           int ldc) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasCgemm] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const hipComplex*, const hipComplex*, int, const hipComplex*, int, const hipComplex*, hipComplex*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasZgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const hipDoubleComplex *alpha, const hipDoubleComplex *A, int lda,
                           const hipDoubleComplex *B, int ldb, const hipDoubleComplex *beta, hipDoubleComplex *C,
                           int ldc) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasZgemm] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const hipDoubleComplex*, const hipDoubleComplex*, int, const hipDoubleComplex*, int, const hipDoubleComplex*, hipDoubleComplex*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, const void *B,
                            hipDataType Btype, int ldb, const void *beta,
                            void *C, hipDataType Ctype, int ldc,
                            hipblasComputeType_t computeType,
                            hipblasGemmAlgo_t algo) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s-m%d-n%d-k%d", __func__, CULiP_get_cublasComputeType_t_string(computeType), m, n , k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}
} // extern "C"
