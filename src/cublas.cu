#include <hipblas.h>
#include <hipblas.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <CULiP/cublas.hpp>
#include "utils.hpp"

#define CULIP_CUBLAS_LIBRARY_NAME       "libcublas.so"
#define CULIP_CUBLAS_ENV_NAME           "CULIP_CUBLAS_LIB_PATH"
#define CULIP_CUBLAS_DISABLE_ENV_NAME   "CULIP_PROFILING_CUBLAS_DISABLE"

extern "C" {
// dlopen cache
void* CULiP_cublas_lib_handle_cache = NULL;

// Control profiling
// 0         = Profiling
// Otherwise = Not profiling
int CULiP_profiling_control_array[CULiP_cublas_enum_length] = {0};

// Controler setter
void CULiP_profile_cublas_enable(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 0;
}
void CULiP_profile_cublas_disable(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 1;
}
void CULiP_profile_cublas_enable_all() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 0;
	}
}
void CULiP_profile_cublas_disable_all() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 1;
	}
}

// hipDataType yo string
#define CULiP_CUBLAS_COMPUTE_T_CASE_STRING(compute_type) case compute_type: return #compute_type
extern "C" const char* CULiP_get_cublasComputeType_t_string(const hipblasComputeType_t compute_type) {
	switch(compute_type) {
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_16F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_16F_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_16F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_FAST_TF32);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32F_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32I);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_32I_PEDANTIC);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_64F);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIPBLAS_COMPUTE_64F_PEDANTIC);
	default:
		break;
	}
	switch((hipDataType)compute_type) {
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_16F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_32F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_32I );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_64F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_8I  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_C_8U  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_16BF);
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_16F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_32F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_32I );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_64F );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_8I  );
		CULiP_CUBLAS_COMPUTE_T_CASE_STRING(HIP_R_8U  );
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasOperation_t_string(const hipblasOperation_t op) {
	switch(op) {
	case HIPBLAS_OP_N:
		return "N";
	case HIPBLAS_OP_T:
		return "T";
	case HIPBLAS_OP_C:
		return "C";
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasFillMode_t_string(const hipblasFillMode_t mode) {
	switch(mode) {
	case HIPBLAS_FILL_MODE_FULL:
		return "FULL";
	case HIPBLAS_FILL_MODE_LOWER:
		return "LOWER";
	case HIPBLAS_FILL_MODE_UPPER:
		return "UPPER";
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasSideMode_t_string(const hipblasSideMode_t mode) {
	switch(mode) {
	case HIPBLAS_SIDE_LEFT:
		return "LEFT";
	case HIPBLAS_SIDE_RIGHT:
		return "RIGHT";
	default:
		return "Unknown";
	}
}

extern "C" const char* CULiP_get_cublasDiagType_t_string(const hipblasDiagType_t type) {
	switch(type) {
	case HIPBLAS_DIAG_NON_UNIT:
		return "NON_UNIT";
	case HIPBLAS_DIAG_UNIT:
		return "UNIT";
	default:
		return "Unknown";
	}
}

// -------------------------------------------------
// GEMM
// -------------------------------------------------

// SGEMM
#define CULIP_FUNC_NAME hipblasSgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemm
#define CULIP_TYPE float
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMM
#define CULIP_FUNC_NAME hipblasDgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemm
#define CULIP_TYPE double
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// HGEMM
#define CULIP_FUNC_NAME hipblasHgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasHgemm
#define CULIP_TYPE half
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMM
#define CULIP_FUNC_NAME hipblasCgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemm
#define CULIP_TYPE hipComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME hipblasZgemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, const void *B,
                            hipDataType Btype, int ldb, const void *beta,
                            void *C, hipDataType Ctype, int ldc,
                            hipblasComputeType_t computeType,
                            hipblasGemmAlgo_t algo) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-%s-m%d-n%d-k%d", __func__, CULiP_get_cublasOperation_t_string(transa), CULiP_get_cublasOperation_t_string(transb), CULiP_get_cublasComputeType_t_string(computeType), m, n , k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

// -------------------------------------------------
// GEMM_BATCHED
// -------------------------------------------------

// SGEMM
#define CULIP_FUNC_NAME hipblasSgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemmBatched
#define CULIP_TYPE float
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMM
#define CULIP_FUNC_NAME hipblasDgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemmBatched
#define CULIP_TYPE double
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// HGEMM
#define CULIP_FUNC_NAME hipblasHgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasHgemmBatched
#define CULIP_TYPE half
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMM
#define CULIP_FUNC_NAME hipblasCgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemmBatched
#define CULIP_TYPE hipComplex
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME hipblasZgemmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemmBatched
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

hipblasStatus_t hipblasGemmBatchedEx(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const void *alpha,
		const void *const Aarray[],
		hipDataType Atype,
		int lda,
		const void *const Barray[],
		hipDataType Btype,
		int ldb,
		const void *beta,
		void *const Carray[],
		hipDataType Ctype,
		int ldc,
		int batchCount,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmBatchedEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void* const[], hipDataType, int, const void* const[], hipDataType, int, const void*, void* const[], hipDataType, int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-%s-m%d-n%d-k%d-batchCount%d", __func__, CULiP_get_cublasOperation_t_string(transa), CULiP_get_cublasOperation_t_string(transb), CULiP_get_cublasComputeType_t_string(computeType), m, n , k, batchCount);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, Aarray, Atype, lda, Barray, Btype, ldb, beta, Carray, Ctype, ldc, batchCount, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

// -------------------------------------------------
// GEMM_STRIDED_BATCHED
// -------------------------------------------------

// SGEMM
#define CULIP_FUNC_NAME hipblasSgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemmStridedBatched
#define CULIP_TYPE float
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMM
#define CULIP_FUNC_NAME hipblasDgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemmStridedBatched
#define CULIP_TYPE double
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// HGEMM
#define CULIP_FUNC_NAME hipblasHgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasHgemmStridedBatched
#define CULIP_TYPE half
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMM
#define CULIP_FUNC_NAME hipblasCgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemmStridedBatched
#define CULIP_TYPE hipComplex
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME hipblasZgemmStridedBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemmBatched
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm_strided_batched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const void *alpha,
		const void *A,
		hipDataType Atype,
		int lda,
		long long int strideA,
		const void *const B,
		hipDataType Btype,
		int ldb,
		long long int strideB,
		const void *beta,
		void *const C,
		hipDataType Ctype,
		int ldc,
		long long int strideC,
		int batchCount,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
	const int profiling_flag = (CULiP_profiling_control_array[CULiP_cublasGemmBatchedEx] == 0) && CULiP_is_profiling_enabled(CULIP_CUBLAS_DISABLE_ENV_NAME);

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer(CULIP_CUBLAS_LIBRARY_NAME, CULIP_CUBLAS_ENV_NAME, __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (profiling_flag) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-%s-m%d-n%d-k%d-batchCount%d", __func__, CULiP_get_cublasOperation_t_string(transa), CULiP_get_cublasOperation_t_string(transb), CULiP_get_cublasComputeType_t_string(computeType), m, n , k, batchCount);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batchCount, computeType, algo);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (profiling_flag) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

// -------------------------------------------------
// GEMV
// -------------------------------------------------

// SGEMV
#define CULIP_FUNC_NAME hipblasSgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgemv
#define CULIP_TYPE float
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGEMV
#define CULIP_FUNC_NAME hipblasDgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgemv
#define CULIP_TYPE double
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGEMV
#define CULIP_FUNC_NAME hipblasCgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemv
#define CULIP_TYPE hipComplex
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMV
#define CULIP_FUNC_NAME hipblasZgemv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemv
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// GBMV
// -------------------------------------------------

// SGBMV
#define CULIP_FUNC_NAME hipblasSgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSgbmv
#define CULIP_TYPE float
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// DGBMV
#define CULIP_FUNC_NAME hipblasDgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDgbmv
#define CULIP_TYPE double
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// CGBMV
#define CULIP_FUNC_NAME hipblasCgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgbmv
#define CULIP_TYPE hipComplex
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGBMV
#define CULIP_FUNC_NAME hipblasZgbmv
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgbmv
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gbmv.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYRK
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsyrk
#define CULIP_TYPE float
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsyrk
#define CULIP_TYPE double
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsyrk
#define CULIP_TYPE hipComplex
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsyrk
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsyrk
#define CULIP_TYPE hipDoubleComplex
#include "cublas.syrk.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYR2K
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsyr2k
#define CULIP_TYPE float
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsyr2k
#define CULIP_TYPE double
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsyr2k
#define CULIP_TYPE hipComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsyr2k
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsyr2k
#define CULIP_TYPE hipDoubleComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYRKX
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsyrkx
#define CULIP_TYPE float
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsyrkx
#define CULIP_TYPE double
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsyrkx
#define CULIP_TYPE hipComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsyrkx
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsyrkx
#define CULIP_TYPE hipDoubleComplex
#include "cublas.syr2k.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// GEMM3M
// -------------------------------------------------
// CGEMM
#define CULIP_FUNC_NAME cublasCgemm3m
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCgemm3m
#define CULIP_TYPE hipComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// ZGEMM
#define CULIP_FUNC_NAME cublasZgemm3m
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZgemm3m
#define CULIP_TYPE hipDoubleComplex
#include "cublas.gemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// SYMM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasSsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasSsymm
#define CULIP_TYPE float
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDsymm
#define CULIP_TYPE double
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCsymm
#define CULIP_TYPE hipComplex
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZsymm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZsymm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.symm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// TRMM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasStrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasStrmm
#define CULIP_TYPE float
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDtrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDtrmm
#define CULIP_TYPE double
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCtrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCtrmm
#define CULIP_TYPE hipComplex
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZtrmm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZtrmm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.trmm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// TRSM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasStrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasStrsm
#define CULIP_TYPE float
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDtrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDtrsm
#define CULIP_TYPE double
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCtrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCtrsm
#define CULIP_TYPE hipComplex
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZtrsm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZtrsm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.trsm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// TRSM Batched
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasStrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasStrsmBatched
#define CULIP_TYPE float
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasDtrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasDtrsmBatched
#define CULIP_TYPE double
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasCtrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasCtrsmBatched
#define CULIP_TYPE hipComplex
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZtrsmBatched
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZtrsmBatched
#define CULIP_TYPE hipDoubleComplex
#include "cublas.trsmBatched.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

// -------------------------------------------------
// HEMM
// -------------------------------------------------

#define CULIP_FUNC_NAME hipblasChemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasChemm
#define CULIP_TYPE hipComplex
#include "cublas.hemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE

#define CULIP_FUNC_NAME hipblasZhemm
#define CULIP_FUNC_ENUM_NAME CULiP_cublasZhemm
#define CULIP_TYPE hipDoubleComplex
#include "cublas.hemm.template.h"
#undef CULIP_FUNC_NAME
#undef CULIP_FUNC_ENUM_NAME
#undef CULIP_TYPE
} // extern "C"
