#include <hipblas.h>
#include <hipblas.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <CULiP/cublas.hpp>
#include "utils.hpp"

extern "C" {
// dlopen cache
void* CULiP_cublas_lib_handle_cache = NULL;

// Control profiling
// 0         = Profiling
// Otherwise = Not profiling
int CULiP_profiling_control_array[CULiP_cublas_enum_length] = {0};

// Controler setter
void CULiP_profile_cublas_enable(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 0;
}
void CULiP_profile_cublas_disable(const CULiP_cublas_control_t target_function) {
	CULiP_profiling_control_array[target_function] = 1;
}
void CULiP_profile_cublas_enable_all() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 0;
	}
}
void CULiP_profile_cublas_disable_all() {
	for (unsigned target_function = 0; target_function < CULiP_cublas_enum_length; target_function++) {
		CULiP_profiling_control_array[target_function] = 1;
	}
}

hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const float*, const float*, int, const float*, int, const float*, float*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer("CULIP_CUBLAS_LIB_PATH", __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (CULiP_profiling_control_array[CULiP_cublasSgemm] == 0) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (CULiP_profiling_control_array[CULiP_cublasSgemm] == 0) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasDgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const double *alpha, const double *A, int lda,
                           const double *B, int ldb, const double *beta, double *C,
                           int ldc) {

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const double*, const double*, int, const double*, int, const double*, double*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer("CULIP_CUBLAS_LIB_PATH", __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (CULiP_profiling_control_array[CULiP_cublasDgemm] == 0) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (CULiP_profiling_control_array[CULiP_cublasDgemm] == 0) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}

hipblasStatus_t hipblasHgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const half *alpha, const half *A, int lda,
                           const half *B, int ldb, const half *beta, half *C,
                           int ldc) {

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const half*, const half*, int, const half*, int, const half*, half*, int);
	*(void**)(&cublas_lib_func) = CULiP_get_function_pointer("CULIP_CUBLAS_LIB_PATH", __func__, &CULiP_cublas_lib_handle_cache);

	hipStream_t hip_stream;
	struct CULiP_profile_result profile_result;

	if (CULiP_profiling_control_array[CULiP_cublasHgemm] == 0) {
		// Get current cuda stream
		hipblasGetStream(handle, &hip_stream);

		// Profile result structure
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-m%d-n%d-k%d", __func__, m, n ,k);

		// Record start rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.start_timestamp);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	if (CULiP_profiling_control_array[CULiP_cublasHgemm] == 0) {
		// Record end rimestamp
		CULiP_launch_function(hip_stream, &CULiP_record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		CULiP_launch_function(hip_stream, &CULiP_print_profile_result, (void*)&profile_result);
	}

	return result;
}
} // extern "C"
