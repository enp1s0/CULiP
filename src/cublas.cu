#include <hipblas.h>
#include <hipblas.h>
#include <dlfcn.h>
#include <iostream>
#include <culibprofiler/cublas.hpp>

extern "C" {
hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {
	return HIPBLAS_STATUS_SUCCESS;
}
}
