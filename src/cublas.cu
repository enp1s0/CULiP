#include <hipblas.h>
#include <hipblas.h>
#include <dlfcn.h>
#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <culibprofiler/cublas.hpp>

#ifdef CULIBPROFILER_ENABLE_DEBUG_PRINT
#define CULIBPROFILER_DEBUG_PRINT(f) (f)
#else
#define CULIBPROFILER_DEBUG_PRINT(f)
#endif

extern "C" {
hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa,
                           hipblasOperation_t transb, int m, int n, int k,
                           const float *alpha, const float *A, int lda,
                           const float *B, int ldb, const float *beta, float *C,
                           int ldc) {
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] start\n", __func__));

	// Get the real library path
	const char* cublas_lib_path = getenv("CLP_CUBLAS_LIB_PATH");
	if (cublas_lib_path == NULL) {
		fprintf(stderr, "[CULiP ERROR] CLP_CUBLAS_LIB_PATH is not set\n");
		exit(1);
	}
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] %s is loaded\n", __func__, cublas_lib_path));

	// Open the library
	void* cublas_lib_handle = dlopen(cublas_lib_path, RTLD_NOW);
	if (cublas_lib_handle == NULL) {
		fprintf(stderr, "[CULiP ERROR] Failed to load the real library %s\n", cublas_lib_path);
		exit(1);
	}

	// Get the function pointer
	hipblasStatus_t (*cublas_lib_func)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const float*, const float*, int, const float*, int, const float*, float*, int);
	*(void**)(&cublas_lib_func) = dlsym(cublas_lib_handle, __func__);
	if (cublas_lib_func == NULL) {
		fprintf(stderr, "[CULiP ERROR] Failed to load the function %s\n", __func__);
		exit(1);
	}

	// Call the function
	const hipblasStatus_t result = (*cublas_lib_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	CULIBPROFILER_DEBUG_PRINT(printf("[CULiP Debug][%s] executed\n", __func__));

	return result;
}
} // extern "C"
